#include "hip/hip_runtime.h"
#include "AKS.h"
#include <fstream>

/*
 * Comparison in GPU between bignumbers
 */
__device__ int gpu_compare(int *a, int *b, int aidx, int bidx)
{
	int i = N - 1;

	while (a[aidx + i] == b[bidx + i] && i > 0)
		i--;
	if (i == N) return 0;
	if (a[aidx + i] > b[bidx + i]) return 1;
	else if (a[aidx + i] == b[bidx + i]) return 0;
	else return -1;
}

/*
 * Multiplication in GPU between bignumbers
 */
__device__ void gpu_multiplication(int *c, int *a, int *b, int aidx, int bidx, int cidx)
{
	int n, s, k;

	// count the digits on each number
	for (k = N - 1; a[aidx + k] == 0; k--);
	n =  k + 1;
	for (k = N - 1; b[bidx + k] == 0; k--);
	s =  k + 1;

	// initialize the result
	int z[2 * N + 1];

	for (int i = 0; i < 2 * N + 1; i++)
		z[i] = 0;

	// calculate the product
	for (int i = 0; i < s; i++)
	{
		int carry = 0;
		for (int j = 0; j < n; j++)
		{
			int temp = z[i + j] + (a[aidx + j] * b[bidx + i]) + carry;
			z[i + j] = temp % 10;
			carry = temp / 10;
		}
		z[n + i] = carry;
	}

	// move the result to the destination
	for (int i = 0; i < N; i++)
		c[cidx + i] = z[i];
}

/*
 * Subtraction in GPU between bignumbers
 */
__device__ void gpu_substraction(int *c, int *a, int *b, int aidx, int bidx, int cidx)
{
	int carry = 0;
	for (int i = 0; i < N; i++)
	{
		int temp = carry + a[aidx + i] - b[bidx + i];
		if (temp < 0)
		{
			// modulo 10
			c[cidx + i] = ((temp % 10) + 10) % 10;
			carry = -1;
		}
		else
		{
			c[cidx + i] = temp % 10;
			carry = 0;
		}
	}
}
/*
 * Addition in GPU between bignumbers
 */
__device__ void gpu_addition(int *c, int *a, int *b, int aidx, int bidx, int cidx)
{
	int carry = 0;
	for (int i = 0; i < N; i++)
	{
		int temp = carry + a[aidx + i] + b[bidx + i];
		c[cidx + i] = temp % 10;
		carry = temp / 10;
	}
}

/*
 * Division in GPU between bignumbers
 */
__device__ void gpu_division(int *c, int *a, int *b, int aidx, int bidx, int cidx)
{
		int temp[N];
		int result[N];
		int x[N];
		for (int i = 0; i < N; i++)
		{
			temp[i] = 0;
			result[i] = 0;
			x[i] = a[aidx + i];
		}
		temp[0] = 1;

		while (gpu_compare(x, b, 0, bidx) != -1)
		{
			gpu_substraction(x, x, b, 0, bidx, 0);
			gpu_addition(result, result, temp, 0, 0, 0);
		}

		for (int i = 0; i < N; i++)
			c[cidx + i] = result[i];
}

/*
 * Modulo in GPU between bignumbers
 */
__device__ void gpu_modulo(int *c, int *a, int *b, int aidx, int bidx, int cidx)
{
	int temp[N];
	int result[N];
	int x[N];
	for (int i = 0; i < N; i++)
	{
		temp[i] = 0;
		result[i] = 0;
		x[i] = a[aidx + i];
	}
	temp[0] = 1;

	while (gpu_compare(x, b, 0, bidx) != -1)
	{
		gpu_substraction(x, x, b, 0, bidx, 0);
		gpu_addition(result, result, temp, 0, 0, 0);
	}
	for (int i = 0; i < N; i++)
		c[cidx + i] = x[i];
}

/*
 * CUDA kernel that calculates the coefficients and
 * checks if they are divible by n
 */
__global__ void calculateCoefficients(int *coeff, int *coeff_max, int *coeff_min)
{
	int ix = threadIdx.x + blockDim.x * blockIdx.x;
	int iy = threadIdx.y + blockDim.y * blockIdx.y;
	// index to third dimension
	int index = ix * 1024 * blockDim.y + iy * 1024;

	int temp[N];
	int res[N];
	int sarray[N];
	int num[N];

	for (int i = 0; i < N; i++)
	{
		temp[i] = 0;
		res[i] = 0;
		sarray[i] = 0;
		num[i] = coeff[index + i];
	}
	temp[0] = 1;
	res[0] = 1;

	//calculate combination
	for (int i = 0; i < ix; i++)
	{
		gpu_substraction(coeff_max, coeff_max, temp, index, 0, index);
		gpu_multiplication(coeff, coeff, coeff_max, index, index, index);
	}

	for (int i = 0; i < ix; i++)
	{
		gpu_addition(coeff_min, coeff_min, temp, index, 0, index);
		gpu_division(coeff, coeff, coeff_min, index, index, index);
	}

	//calculate power
	int exp = ix + 1;
	int s = iy + 1;
	int j = 0;
	while(s > 0)
	{
		sarray[j++] = s % 10;
		s = s / 10;
	}

	for (int i = 0; i < exp; i++)
	{
		gpu_multiplication(res, res, sarray, 0, 0, 0);
		gpu_modulo(res, res, num, 0, 0, 0);
	}
	gpu_modulo(res, res, num, 0, 0, 0);
	//calculate modulo

	gpu_multiplication(coeff, coeff, res, index, 0, index);
	gpu_modulo(coeff, coeff, num, index, 0, index);

	//if everything is zero return true;
}

/*
 * AKS Bernstein using CUDA
 */
bool AKS::AKSAlgorithm_Bernstein_CUDA()
{
	struct timeval start, end;
	gettimeofday(&start, NULL);
	if (pp.AlgorithmX(number))
	{
		gettimeofday(&end, NULL);
		std::cout << "Elapsed time (perfect power) = " << ((end.tv_sec  - start.tv_sec) * 1000000u + end.tv_usec - start.tv_usec) / 1.e6 << std::endl;
		return false;
	}
	gettimeofday(&end, NULL);
	std::cout << "Elapsed time (perfect power) = " << ((end.tv_sec  - start.tv_sec) * 1000000u + end.tv_usec - start.tv_usec) / 1.e6 << std::endl;

	gettimeofday(&start, NULL);
	int r = primitiveModulo();
	gettimeofday(&end, NULL);
	std::cout << "r = " << r << std::endl;
	std::cout << "Elapsed time (find r) = " << ((end.tv_sec  - start.tv_sec) * 1000000u + end.tv_usec - start.tv_usec) / 1.e6 << std::endl;

	int phiR = phi(NTL::ZZ(r));
	const double d = 0.5 * phiR;
	const double i = 0.475 * phiR;
	const double j = 0.475 * phiR;

	gettimeofday(&start, NULL);
	long s = findSB(r, d, i, j);
	std::cout << "s = " << s <<std::endl;
	gettimeofday(&end, NULL);
	std::cout << "Elapsed time (find s) = " << ((end.tv_sec  - start.tv_sec) * 1000000u + end.tv_usec - start.tv_usec) / 1.e6 << std::endl;
	bool result = true;

	gettimeofday(&start, NULL);
	#pragma omp parallel for reduction(&:result)
	for (int i = 2; i <= s + 1; i++)
	{
		NTL::ZZ iZZ(i);
		if ((iZZ % number != 0) && (NTL::PowerMod(iZZ % number, number - 1, number) != 1))
			result &= false;
//		if ((i != number) && NTL::GCD(number, i) != 1)
//			return false;
		for (int j = 2; j <= s + 1; j++)
		{
			NTL::ZZ jZZ(j);
			if ((iZZ * jZZ > 1) && ((iZZ * jZZ) - 1 != number) && (iZZ * jZZ - 1 < number) && (NTL::GCD(number, (iZZ * jZZ) - 1) != 1))
				result &= false;
			if ((iZZ - jZZ > 0) && (iZZ-jZZ != number) && (iZZ-jZZ < number) && (NTL::GCD(number, iZZ - jZZ) != 1))
				result &= false;
		}
	}
	gettimeofday(&end, NULL);
	if (result == false)
		return false;
	std::cout << "Elapsed time (gcd) = " << ((end.tv_sec  - start.tv_sec) * 1000000u + end.tv_usec - start.tv_usec) / 1.e6 << std::endl;

	gettimeofday(&start, NULL);

	// initialise the memory for the host
	int int_num = NTL::conv<int>(number);
	// the calculated coefficients. Initialized as h_coeff = n
	int h_coeff[int_num - 1][s][N];
	// helps to calculate combination
	int h_coeff_max[int_num - 1][s][N];
	int h_coeff_min[int_num - 1][s][N];

	// initialise
	for (int i = 0; i < int_num - 1; i++)
		for (int j = 0; j < s; j++)
			for (int z = 0; z < N; z++)
			{
				h_coeff[i][j][z] = num_arr[N - z - 1];
				h_coeff_max[i][j][z] = num_arr[N - z - 1];
				if(z == 0)
					h_coeff_min[i][j][z] = 1;
				else
					h_coeff_min[i][j][z] = 0;
			}

	// initialize the memory for the device
	int *d_coeff;
	int *d_coeff_max;
	int *d_coeff_min;
	hipError_t err;

	// Malloc copy on the device
	err = hipMalloc((void**) &d_coeff, s * (int_num - 1) * N * sizeof(int));
	std::cout << hipGetErrorString(err) << std::endl;
	err = hipMalloc((void**) &d_coeff_max, s * (int_num - 1) * N * sizeof(int));
	std::cout << hipGetErrorString(err) << std::endl;
	err = hipMalloc((void**) &d_coeff_min, s * (int_num - 1) * N * sizeof(int));
	std::cout << hipGetErrorString(err) << std::endl;

	// transfer to device memory
	err = hipMemcpy(d_coeff, h_coeff, s * (int_num - 1) * N * sizeof(int), hipMemcpyHostToDevice);
	std::cout << hipGetErrorString(err) << std::endl;
	err = hipMemcpy(d_coeff_max, h_coeff_max, s * (int_num - 1) * N * sizeof(int), hipMemcpyHostToDevice);
	std::cout << hipGetErrorString(err) << std::endl;
	err = hipMemcpy(d_coeff_min, h_coeff_min, s * (int_num - 1) * N * sizeof(int), hipMemcpyHostToDevice);
	std::cout << hipGetErrorString(err) << std::endl;

	// build the geometry of the problem
	dim3 block;
	dim3 grid;
	if ((int_num - 1) * s > 1024)
	{
		block.x = 32;
		block.y = 32;
		grid.x = (int_num - 1) / 1024;
		grid.y = s / 1024;
	}
	else
	{
		block.x = int_num - 1;
		block.y = s;
		grid.x = 1;
	}

	// calling the kernel
	calculateCoefficients<<<grid, block>>>(d_coeff, d_coeff_max, d_coeff_min);

	// get the result from device to host
	err = hipMemcpy(h_coeff, d_coeff, s * (int_num - 1) * N * sizeof(int), hipMemcpyDeviceToHost);
	std::cout << hipGetErrorString(err) << std::endl;

	// check the result
	for (int i = 0; i < int_num - 1; i++)
		for (int j = 0; j < s; j++)
		{
			for (int z = 0; z < N; z++)
			{
				if (h_coeff[i][j][z] != 0)
					return false;
			}
		}

	// free memory
	hipFree(d_coeff);
	hipFree(d_coeff_max);
	hipFree(d_coeff_min);

	gettimeofday(&end, NULL);
	std::cout << "Elapsed time (congruence) = " << ((end.tv_sec  - start.tv_sec) * 1000000u + end.tv_usec - start.tv_usec) / 1.e6 << std::endl;

	return true;
}
